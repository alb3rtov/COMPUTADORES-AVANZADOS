/* Copiar traspuesta de matriz h_a[F][C] en matriz h_b[C][F] aunque el n.º de hebras de 
   los bloques no divida al n.º de componentes de las matrices */
// #include <stdlib.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

#define F 25
#define C 43
// matriz original de F filas y C columnas
#define H 8
#define K 8
// bloques de H x K hebras (HxK<=512, cap. cpto. 1.3)

 __global__ void trspta1coalesc(int *dev_a, int *dev_b, int filas, int cols)
{
  __shared__ int tile[H][K];

  int x_index = blockIdx.x * blockDim.x;	
  int y_index = blockIdx.y * blockDim.y;

  int ix = x_index + threadIdx.x;
  int iy = y_index + threadIdx.y;

  if ((ix<cols)&&(iy<filas)) {
	  tile[threadIdx.y][threadIdx.x] = dev_a[ix+cols*iy];
  }	  

  __syncthreads();	
  
  int index_out = x_index * filas + y_index;
  
  if (((x_index+threadIdx.y)<cols) && ((y_index+threadIdx.x)<filas)) {
	  dev_b[index_out + threadIdx.y * filas + threadIdx.x] = tile[threadIdx.x][threadIdx.y];
  }
}

int main(int argc, char** argv)
{
  int h_a[F][C], h_b[C][F];
  int *d_a, *d_b;
  int i, j, aux, size = F * C * sizeof(int);
  dim3 hebrasBloque(H, K); // bloques de H x K hebras
  int numBlf = (F+H-1)/H;  // techo de F/H
  int numBlc = (C+K-1)/K;  // techo de C/K
  dim3 numBloques(numBlc,numBlf);

  // reservar espacio en el device para d_a y d_b
  hipMalloc((void**) &d_a, size); 
  hipMalloc((void**) &d_b, size);

  // dar valores a la matriz h_a en la CPU e imprimirlos
  printf("\nMatriz origen\n");
  for (i=0; i<F; i++) {
    for (j=0; j<C; j++) {
      aux = i*C+j;
      h_a[i][j] = aux;
      printf("%d ", aux);
    }
    printf("\n");
  }

  // copiar matriz h_a en d_a
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  
  // llamar al kernel que obtiene en d_b la traspuesta de d_a
  trspta1coalesc<<<numBloques, hebrasBloque>>>(d_a, d_b, F, C);

  // copiar matriz d_b en h_b
  hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  for (i=0; i<F; i++)
    for (j=0; j<C; j++) 
      if (h_a[i][j]!= h_b[j][i]) 
		{printf("error en componente %d %d de matriz de entrada \n", i,j); break;}
 
// imprimir matriz resultado
  printf("\nMatriz resultado\n");
  for (i=0; i<C; i++) {
    for (j=0; j<F; j++) {
      printf("%d ", h_b[i][j]);
    }
    printf("\n");
  }
  printf("\n");

  hipFree(d_a); hipFree(d_b);
  
  return 0;
} 
