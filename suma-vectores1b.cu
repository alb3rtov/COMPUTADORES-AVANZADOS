
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 600 //En mi caso N debe ser mayor a 1024 para que de un error

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int i = threadIdx.x;
  DC[i] = DA[i] + DB[i];
}

int main()
{ int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;
  int i; int size = N*sizeof(int);
  
  hipError_t errSync;
  hipError_t errAsync;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // reservamos espacio en la memoria global del device
  
  hipMalloc((void**)&DA, size);
  
  errAsync = hipDeviceSynchronize();
  if (errAsync != hipSuccess)
    printf("Async malloc error: %s\n", hipGetErrorString(errAsync));

  hipMalloc((void**)&DB, size);
  
  errAsync = hipDeviceSynchronize();
  if (errAsync != hipSuccess)
    printf("Async malloc error: %s\n", hipGetErrorString(errAsync));

  hipMalloc((void**)&DC, size);

  errAsync = hipDeviceSynchronize();
  if (errAsync != hipSuccess)
    printf("Async malloc error: %s\n", hipGetErrorString(errAsync));

  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  
  hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  
  errSync = hipGetLastError();
  if (errSync != hipSuccess)
    printf("Sync memcpy error: %s\n", hipGetErrorString(errSync));
  

  hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
    
  errSync = hipGetLastError();
  if (errSync != hipSuccess)
    printf("Sync memcpy error: %s\n", hipGetErrorString(errSync));

  // llamamos al kernel (1 bloque de N hilos)
  hipEventRecord(start);
  VecAdd <<<1, N>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Total sesion time (ms): %f\n", milliseconds);

  //cudaEventRecord(start);
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  
  errSync = hipGetLastError();

  if (errSync != hipSuccess)
    printf("Sync memcpy error: %s\n", hipGetErrorString(errSync));

  errSync = hipGetLastError();
  errAsync = hipDeviceSynchronize();

  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

  // liberamos la memoria reservada en el device
  hipFree(DA); 
  hipFree(DB); 
  hipFree(DC);

  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{printf("error en componente %d\n", i); break;}

  return 0;
} 
