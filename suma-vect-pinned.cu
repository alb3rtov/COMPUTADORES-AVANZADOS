
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100000
#define tb 512	// tamaño bloque

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	int ii = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i=ii; i<N; i+=stride)
	    DC[i] = DA[i] + DB[i];
}

int main()
{ hipFree(0);
  //int HA[N], HB[N], HC[N];
  int *HA, *HB, *HC, *DA, *DB, *DC;
  int i, dg; int size = N*sizeof(int);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipError_t testerr;
  testerr = hipHostMalloc((void**)&HA, size, hipHostMallocDefault);
  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc HA: %s\n", hipGetErrorString(testerr));
	exit(0);
  }	
  testerr = hipHostMalloc((void**)&HB, size, hipHostMallocDefault);
  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc HB: %s\n", hipGetErrorString(testerr));
	exit(0);
  }		
  testerr = hipHostMalloc((void**)&HC, size, hipHostMallocDefault);
  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc HC: %s\n", hipGetErrorString(testerr));		
	exit(0);
  }

  // reservamos espacio en la memoria global del device
  
  testerr = hipHostMalloc((void**)&DA, size, hipHostMallocDefault);  

  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc DA: %s\n", hipGetErrorString(testerr));
	exit(0);
  }
  
  testerr = hipHostMalloc((void**)&DB, size, hipHostMallocDefault);
  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc DB: %s\n", hipGetErrorString(testerr));
	exit(0);
  }

  //cudaEventRecord(start);
  testerr = hipHostMalloc((void**)&DC, size, hipHostMallocDefault);
  //cudaEventRecord(stop);
  //cudaEventSynchronize(stop);
  //float milliseconds = 0;
  //cudaEventElapsedTime(&milliseconds, start, stop);

  //printf("DB (ms): %f\n", milliseconds);

  if (testerr!= hipSuccess) {
	printf("Error en hipMalloc DC: %s\n", hipGetErrorString(testerr));		
	exit(0);
  }
    
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  testerr = hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  if (testerr != hipSuccess) {
	printf("Error en hipMemcpy del host al device: %s\n", hipGetErrorString(testerr));		
	exit(0);
  }
  testerr = hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  if (testerr != hipSuccess) {
	printf("Error en hipMemcpy del host al device: %s\n", hipGetErrorString(testerr));		
	exit(0);
  }      
      
  dg = (N+tb-1)/tb; if (dg>65535) dg=65535;
  // llamamos al kernel
  VecAdd <<<dg, tb>>>(DA, DB, DC);	// N o más hilos ejecutan el kernel en paralelo
  testerr = hipGetLastError();
  if (testerr!= hipSuccess) {
    printf("Error al ejecutar el kernel: %s\n", hipGetErrorString(testerr));
	exit(0);
  }    
  
  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  testerr = hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  if (testerr != hipSuccess) {
	printf("Error en hipMemcpy del device al host: %s\n", hipGetErrorString(testerr));		
	exit(0);  
  }
   
  // liberamos la memoria reservada en el device
  hipFree(DA); hipFree(DB); hipFree(DC);  
  //cudaFreeHost(HA); cudaFreeHost(HB); cudaFreeHost(HC);  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{printf("error en componente %d\n", i); break;}
  
  return 0;
} 
